#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * \file Externally defined CUDA kernels for use in TVM runtime
 */

#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <dlpack/dlpack.h>
#include <tvm/runtime/registry.h>

#include "../../cuda/cuda_common.h"

namespace tvm {
namespace contrib {

using namespace runtime;

template <typename T>
__device__ T device_max(T a, T b) {
  return max(a, b);
}

template <>
__device__ __half device_max(__half a, __half b) {
  return __hmax(a, b);
}

template <typename T>
__device__ T device_abs(T a) {
  return abs(a);
}

template <>
__device__ __half device_abs(__half a) {
  return __habs(a);
}

template <typename T>
__inline__ __device__ T warp_reduce_max(T val) {
  for (int offset = warpSize / 2; offset > 0; offset >>= 1) {
    val = device_max(val, __shfl_down_sync(0xffffffff, val, offset));
  }
  return val;
}

// Single block reduce, assumes size % 1024 == 0
template <typename T>
__global__ void max_reduce_kernel_single_block(T* input, T* output, int size) {
  __shared__ T shared[32];

  int tid = threadIdx.x;
  T max_val = std::numeric_limits<T>::lowest();

  // Step 1: Each thread reduces across the elements it owns
  for (int i = tid; i < size; i += blockDim.x) {
    // use __hmax for float16
    max_val = device_max(max_val, device_abs(input[i]));
  }

  // Step 2: Perform reduce across warps
  max_val = warp_reduce_max(max_val);

  // Step 3: Write the reduced value from each warp to shared memory
  if (tid % warpSize == 0) {
    shared[tid / warpSize] = max_val;
  }
  __syncthreads();

  // Step 4: Perform a final reduction in the first warp across shared values
  if (tid < warpSize) {
    max_val = shared[tid];
    max_val = warp_reduce_max(max_val);
    if (tid == 0) {
      *output = max_val;
    }
  }
}

template __global__ void max_reduce_kernel_single_block<float>(float* input, float* output,
                                                               int size);
template __global__ void max_reduce_kernel_single_block<__half>(__half* input, __half* output,
                                                                int size);
template <typename T>
void LaunchMaxReduceKernelSingleBlock(DLTensor* input, DLTensor* output, int size) {
  T* input_ptr = static_cast<T*>(input->data);
  T* output_ptr = static_cast<T*>(output->data);

  int blocks = 1;
  int threads = 1024;
  max_reduce_kernel_single_block<T><<<blocks, threads>>>(input_ptr, output_ptr, size);
}

TVM_REGISTER_GLOBAL("tvm.contrib.cuda.reduce_max_abs").set_body([](TVMArgs args, TVMRetValue* ret) {
  DLTensor* input = args[0];
  DLTensor* output = args[1];

  int size = 1;
  for (int i = 0; i < input->ndim; ++i) {
    size *= input->shape[i];
  }

  CHECK_EQ(size % 1024, 0) << "tvm.contrib.cuda.reduce_max_abs currently only supports reducing "
                              "tensors that are an even factor of 1024 elements";

  auto dtype = DLDataType2String(input->dtype);

  if (dtype == "float32") {
    LaunchMaxReduceKernelSingleBlock<float>(input, output, size);
  } else if (dtype == "float16") {
    LaunchMaxReduceKernelSingleBlock<__half>(input, output, size);
  } else {
    LOG(FATAL) << "Unsupported input dtype: " << dtype;
  }
});

}  // namespace contrib
}  // namespace tvm
